#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    const int N = 512;
    int size = N * sizeof(int);
    int *h_a = (int *)malloc(size);
    int *h_b = (int *)malloc(size);
    int *h_c = (int *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    add<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
